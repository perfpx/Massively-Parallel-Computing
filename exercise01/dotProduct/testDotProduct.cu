// ==========================================================================
// $Id$
// ==========================================================================
// (C)opyright: 2009-2010
//
//   Ulm University
//
// Creator: Hendrik Lensch
// Email:   {hendrik.lensch,johannes.hanika}@uni-ulm.de
// ==========================================================================
// $Log$
// ==========================================================================

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

using namespace std;

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

#define MAX_BLOCKS 256
#define MAX_THREADS 128

#define RTEST // use random initialization of array

/* compute the dot product between a1 and a2. a1 and a2 are of size
 dim. The result of each thread should be stored in _dst[blockIdx.x *
 blockDim.x + threadIdx.x]. Each thread should accumulate the dot
 product of a subset of elements.
 */
__global__ void dotProdKernel(float* _dst, const float* _a1, const float* _a2, int _dim)
{

    // program your kernel here
    //!!!!!!!!! missing  !!!!!!!!!!!!!!!!!!!!!!!!
}

/* This program sets up two large arrays of size dim and computes the
dot product of both arrays.

The arrays are uploaded only once and the dot product is computed
multiple times. While this does not make too much sense it
demonstrated the possible speedup.  */
int main(int argc, char* argv[])
{
    // parse command line
    int acount = 1;

    if (argc < 3)
    {
        printf("usage: testDotProduct <dim> <GPU-flag [0,1]>\n");
        exit(1);
    }

    // number of elements in both vectors
    int dim = atoi(argv[acount++]);

    // flag indicating weather the CPU or the GPU version should be executed
    bool gpuVersion = atoi(argv[acount++]);

    printf("dim: %d\n", dim);

    float* cpuArray1 = new float[dim];
    float* cpuArray2 = new float[dim];

    // initialize the two arrays (either random or deterministic)
    for (int i = 0; i < dim; ++i)
    {
#ifdef RTEST
        cpuArray1[i] = drand48();
        cpuArray2[i] = drand48();
#else
        cpuArray1[i] = 2.0;
        cpuArray2[i] = i % 10;
#endif
    }

    // now the gpu stuff
    float* gpuArray1;
    float* gpuArray2;
    float* gpuResult;

    float* h;

    if (gpuVersion)
    {
        // allocate two gpuArray 1 and gpuArray 2 and gpuResult array on GPU

        //!!!!!!!!! missing  !!!!!!!!!!!!!!!!!!!!!!!!

        // copy the array once to the device

        //!!!!!!!!! missing  !!!!!!!!!!!!!!!!!!!!!!!!

        // allocate an array to download the results of all threads
        h = new float[MAX_BLOCKS * MAX_THREADS];
    }

    const int num_iters = 100;
    double finalDotProduct;

    if (!gpuVersion)
    {
        printf("cpu: ");
        for (int iter = 0; iter < num_iters; ++iter)
        {
            finalDotProduct = 0.0;
            for (int i = 0; i < dim; ++i)
            {
                finalDotProduct += cpuArray1[i] * cpuArray2[i];
            }
        }
    }
    else
    {

        // CUDA version here
        printf("gpu: ");

        // a simplistic way of splitting the problem into threads
        dim3 blockGrid(MAX_BLOCKS);
        dim3 threadBlock(MAX_THREADS);

        for (int iter = 0; iter < num_iters; ++iter)
        {
            dotProdKernel<<<blockGrid, threadBlock>>>(gpuResult, gpuArray1, gpuArray2, dim);
        }

        // download and combine the results of multiple threads on the CPU

        //!!!!!!!!! missing  !!!!!!!!!!!!!!!!!!!!!!!!
    }

    printf("Result: %f\n", finalDotProduct);

    if (gpuVersion)
    {

        // cleanup GPU memory

        //!!!!!!!!! missing  !!!!!!!!!!!!!!!!!!!!!!!!

        delete[] h;
    }

    delete[] cpuArray2;
    delete[] cpuArray1;

    checkCUDAError("end of program");

    printf("done\n");
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}
