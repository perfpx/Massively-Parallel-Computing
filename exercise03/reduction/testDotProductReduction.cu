// ==========================================================================
// $Id$
// ==========================================================================
// (C)opyright: 2009
//
//   Ulm University
//
// Creator: Hendrik Lensch, Holger Dammertz
// Email:   hendrik.lensch@uni-ulm.de, holger.dammertz@uni-ulm.de
// ==========================================================================
// $Log$
// ==========================================================================

#include <algorithm>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <ctime>
#include <hip/hip_runtime.h>
#include <sys/time.h>

using namespace std;

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

#define MAX_BLOCKS 256
#define MAX_THREADS 256

inline __int64_t continuousTimeNs()
{
    timespec now;
    clock_gettime(CLOCK_REALTIME, &now);

    __int64_t result = (__int64_t)now.tv_sec * 1000000000 + (__int64_t)now.tv_nsec;

    return result;
}

__global__ void dotProdKernel(float* dst, const float* a1, const float* a2, int dim)
{
    // Number of the current thread
    unsigned int threadNo = blockDim.x * blockIdx.x + threadIdx.x;
    // Number of all threads
    unsigned int threadSize = gridDim.x * blockDim.x;

    // Sum up every (threadSize)th element starting at the threads index and ending before dim
    float result = 0.0f;
    for (unsigned int t = threadNo; t < dim; t += threadSize)
        result += a1[t] * a2[t];

    // Write the result to dst[threadIdx] if it can contain something
    dst[threadNo] = result;
}

// !!! missing !!!
// Kernel for reducing gridDim.x*blockDim.x elements to gridDim.x elements

/* This program sets up two large arrays of size dim and computes the
 dot product of both arrays.

 Most of the code of previous exercises is reused.
 Mode 0 of the program computes the final dot product as before.

 Mode 1: After computing the dot product and storing the result for all
 MAX_BLOCKS * MAX_THREAD threads, this time, the reduction of the sum
 is to be computed on the GPU.

 Write a reduction sum kernel which reduces the input to the sum in log(n) steps.
 The number of total threads will be divided by nThreads(iter-1) in each iteration.

 Inside the kernel, the problem will be reduced by a factor of 2 in each step.

 */

int main(int argc, char* argv[])
{

    // parse command line
    int acount = 1;

    if (argc < 3)
    {
        printf("usage: testDotProductStreams <dim> <reduction mode [CPU only:0, CPU sum:1, GPU reduction:2]>\n");
        exit(1);
    }

    // number of elements in both vectors
    int dim = atoi(argv[acount++]);

    int mode = atoi(argv[acount++]);

    printf("dim: %d\n", dim);

    // Allocate only pagelocked memory for simplicity
    float* cpuArray1;
    float* cpuArray2;
    float* cpuResult;
    hipHostMalloc((void**)&cpuArray1, dim * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void**)&cpuArray2, dim * sizeof(float), hipHostMallocDefault);
    hipHostMalloc((void**)&cpuResult, MAX_THREADS * MAX_BLOCKS * sizeof(float), hipHostMallocDefault);

    // initialize the two arrays
    for (int i = 0; i < dim; ++i)
    {
#ifdef RTEST
        cpuArray1[i] = drand48();
        cpuArray2[i] = drand48();
#else
        cpuArray1[i] = 1.0;
        cpuArray2[i] = 1.; // i % 10;
#endif
    }

    // Allocate GPU memory
    float* gpuArray1;
    float* gpuArray2;
    float* gpuResult1; // Two result arrays to be able to move data from one to the other during
                       // reduction
    float* gpuResult2;
    hipMalloc((void**)&gpuArray1, dim * sizeof(float));
    hipMalloc((void**)&gpuArray2, dim * sizeof(float));

    hipMalloc((void**)&gpuResult1, MAX_BLOCKS * MAX_THREADS * sizeof(float));

    hipMalloc((void**)&gpuResult2,
               MAX_BLOCKS * MAX_THREADS
                   * sizeof(float)); // MAX_BLOCKS elements would be sufficient here...

    // Upload input data

    hipMemcpy(gpuArray1, cpuArray1, dim * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(gpuArray2, cpuArray2, dim * sizeof(float), hipMemcpyHostToDevice);

    // Variable for output
    double finalDotProduct = 0.;

    __int64_t startTime = continuousTimeNs();

    // Iterations for benchmarking only the kernel call
    for (int iter = 0; iter < 1000; ++iter)
    {
        // a simplistic way of splitting the problem into threads
        dim3 blockGrid(MAX_BLOCKS);
        dim3 threadBlock(MAX_THREADS);

        unsigned int expectedResultSize;

        switch (mode)
        {
        case 0:
            finalDotProduct = 0.0;

            for (unsigned int i = 0; i < dim; i++)
                finalDotProduct += cpuArray1[i] * cpuArray2[i];

            break;

        case 1:
            // call the dot kernel
            dotProdKernel<<<blockGrid, threadBlock>>>(gpuResult1, gpuArray1, gpuArray2, dim);

            // If dim < launchedThreads, only the first dim elements will contain data
            expectedResultSize = min(dim, MAX_THREADS * MAX_BLOCKS);

            // download and combine the results of multiple threads

            hipMemcpy(cpuResult, gpuResult1, expectedResultSize * sizeof(float),
                       hipMemcpyDeviceToHost);

            finalDotProduct = 0.;

            // accumulate the final result on the host
            for (int i = 0; i < expectedResultSize; ++i)
                finalDotProduct += cpuResult[i];

            break;

        case 2:
            // call the dot kernel, store result in gpuResult1
            dotProdKernel<<<blockGrid, threadBlock>>>(gpuResult1, gpuArray1, gpuArray2, dim);

            // !!! missing !!!
            // Reduce all the dot product summands to one single value,
            // download it to a float and use it to set finalDotProduct.

            break;

        } // end switch
    }

    __int64_t endTime = continuousTimeNs();
    __int64_t runTime = endTime - startTime;

    // Print results and timing
    printf("Result: %f\n", finalDotProduct);
    printf("Time: %f\n", (float)runTime / 1000000000.0f);

    // cleanup GPU memory
    hipFree(gpuResult1);
    hipFree(gpuResult2);
    hipFree(gpuArray2);
    hipFree(gpuArray1);

    // free page locked memory
    hipHostFree(cpuArray1);
    hipHostFree(cpuArray2);
    hipHostFree(cpuResult);

    checkCUDAError("end of program");

    printf("done\n");
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}
