#include "hip/hip_runtime.h"
// ==========================================================================
// $Id$
// ==========================================================================
// (C)opyright: 2009
//
//   Ulm University
//
// Creator: Hendrik Lensch, Holger Dammertz
// Email:   hendrik.lensch@uni-ulm.de, holger.dammertz@uni-ulm.de
// ==========================================================================
// $Log$
// ==========================================================================

#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <vector>

#include "PPM.hh"

using namespace std;
using namespace ppm;

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

__device__ __constant__ float3 gpuClusterCol[2048];

#define THREADS 256
#define LOG_IMG_SIZE 8
#define IMG_SIZE 256
#define WINDOW 6

/* The function measures for every pixel the distance to all
 clusters, and determines the clusterID of the nearest cluster
 center. It then colors the pixel in the cluster's color.

 The cluster centers are given as an array of linear indices into
 the vector image, i.e.    _clusterInfo[0] = (x_0 + y_0 * _w).

 */
__global__ void voronoiKernel(float3* _dst, int _w, int _h, int _nClusters, const int* _clusterInfo)
{
    // get the shared memory
    extern __shared__ int shm[];

    int nIter = _nClusters / THREADS + 1;
    // load cluster data
    for (int i = 0; i < nIter; ++i)
    {
        int pos = i * THREADS + threadIdx.x;
        if (pos < _nClusters)
        {
            shm[pos] = _clusterInfo[pos];
        }
    }

    __syncthreads();

    // compute the position within the image
    float x = blockIdx.x * blockDim.x + threadIdx.x;
    float y = blockIdx.y;

    int pos = x + y * _w;

    // determine which is the closest cluster
    float minDist = 1000000.;
    int minIdx = 0;
    for (int i = 0; i < _nClusters; ++i)
    {

        float yy = shm[i] >> LOG_IMG_SIZE;
        float xx = shm[i] % IMG_SIZE;

        float dist = (x - xx) * (x - xx) + (y - yy) * (y - yy);
        if (dist < minDist)
        {
            minDist = dist;
            minIdx = i;
        }
    }

    _dst[pos].x = gpuClusterCol[minIdx].x;
    _dst[pos].y = gpuClusterCol[minIdx].y;
    _dst[pos].z = gpuClusterCol[minIdx].z;

    // mark the center of each cluster
    if (minDist <= 2.)
    {
        _dst[pos].x = 255;
        _dst[pos].y = 0.;
        _dst[pos].z = 0.;
    }
}

__device__ float luminance(const float4& _col)
{
    return 0.299 * _col.x + 0.587 * _col.y + 0.114 * _col.z;
}

/** stores a 1 in _dst if the pixel's luminance is a maximum in the
WINDOW x WINDOW neighborhood
 */
__global__ void featureKernel(int* _dst, hipTextureObject_t texImg, int _w, int _h)
{
    // compute the position within the image
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y;

    float lum = luminance(tex2D<float4>(texImg, x, y));

    bool maximum = false;

    if (lum > 20)
    {
        maximum = true;
        for (int v = y - WINDOW; v < y + WINDOW; ++v)
        {
            for (int u = x - WINDOW; u < x + WINDOW; ++u)
            {

                if (lum < luminance(tex2D<float4>(texImg, u, v)))
                {
                    maximum = false;
                }
            }
        }
    }

    if (maximum)
    {
        _dst[x + y * _w] = 1;
    }
    else
    {
        _dst[x + y * _w] = 0;
    }
}

// !!! missing !!!
// Kernels for Prefix Sum calculation (compaction, spreading, possibly shifting)
// and for generating the gpuFeatureList from the prefix sum.

/* This program detects the local maxima in an image, writes their
location into a vector and then computes the Voronoi diagram of the
image given the detected local maxima as cluster centers.

A Voronoi diagram simply colors every pixel with the color of the
nearest cluster center. */

int main(int argc, char* argv[])
{

    // parse command line
    int acount = 1;
    if (argc < 4)
    {
        printf("usage: testPrefix <inImg> <outImg> <mode>\n");
        exit(1);
    }
    string inName(argv[acount++]);
    string outName(argv[acount++]);
    int mode = atoi(argv[acount++]);

    // Load the input image
    float* cpuImage;
    int w, h;
    readPPM(inName.c_str(), w, h, &cpuImage);
    int nPix = w * h;

    // Allocate GPU memory
    int* gpuFeatureImg; // Contains 1 for a feature, 0 else
    // Can be used to do the reduction step of prefix sum calculation in place
    int* gpuPrefixSumShifted; // Output buffer containing the prefix sum
    // Shifted by 1 since it contains 0 as first element by definition
    int* gpuFeatureList; // List of pixel indices where features can be found.
    float3* gpuVoronoiImg; // Final rgb output image
    hipMalloc((void**)&gpuFeatureImg, (nPix) * sizeof(int));

    hipMalloc((void**)&gpuPrefixSumShifted, (nPix + 1) * sizeof(int));
    hipMalloc((void**)&gpuFeatureList, 10000 * sizeof(int));

    hipMalloc((void**)&gpuVoronoiImg, nPix * 3 * sizeof(float));

    // color map for the cluster
    float clusterCol[2048 * 3];
    float* ci = clusterCol;
    for (int i = 0; i < 2048; ++i, ci += 3)
    {
        ci[0] = 32 * i % 256;
        ci[1] = (10 * i + 128) % 256;
        ci[2] = (40 * i + 255) % 256;
    }

    hipMemcpyToSymbol(HIP_SYMBOL(gpuClusterCol), clusterCol, 2048 * 3 * sizeof(float));

    hipArray* gpuTex;
    hipChannelFormatDesc floatTex = hipCreateChannelDesc<float4>();
    hipMallocArray(&gpuTex, &floatTex, w, h);

    // pad to float4 for faster access
    float* img4 = new float[w * h * 4];

    for (int i = 0; i < w * h; ++i)
    {
        img4[4 * i] = cpuImage[3 * i];
        img4[4 * i + 1] = cpuImage[3 * i + 1];
        img4[4 * i + 2] = cpuImage[3 * i + 2];
        img4[4 * i + 3] = 0.;
    }

    // upload to array

    hipMemcpy2DToArray(gpuTex, 0, 0, img4, w * 4 * sizeof(float), w * 4 * sizeof(float), h,
                        hipMemcpyHostToDevice);

    // create texture object
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = gpuTex;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType;

    hipTextureObject_t tex = 0;
    hipCreateTextureObject(&tex, &resDesc, &texDesc, nullptr);

    cout << "setup texture" << endl;
    cout.flush();

    // calculate the block dimensions
    dim3 threadBlock(THREADS);
    dim3 blockGrid(w / THREADS, h, 1);

    printf("blockDim: %d  %d \n", threadBlock.x, threadBlock.y);
    printf("gridDim: %d  %d \n", blockGrid.x, blockGrid.y);

    featureKernel<<<blockGrid, threadBlock>>>(gpuFeatureImg, tex, w, h);

    // variable to store the number of detected features = the number of clusters
    int nFeatures;

    if (mode == 0)
    {
        ////////////////////////////////////////////////////////////
        // CPU compaction:
        ////////////////////////////////////////////////////////////

        // download result

        hipMemcpy(cpuImage, gpuFeatureImg, nPix * sizeof(float), hipMemcpyDeviceToHost);

        std::vector<int> features;

        float* ii = cpuImage;
        for (int i = 0; i < nPix; ++i, ++ii)
        {
            if (*ii > 0)
            {
                features.push_back(i);
            }
        }

        cout << "nFeatures: " << features.size() << endl;

        nFeatures = features.size();
        // upload feature vector

        hipMemcpy(gpuFeatureList, &(features[0]), nFeatures * sizeof(int), hipMemcpyHostToDevice);
    }
    else
    {
        ////////////////////////////////////////////////////////////
        // GPU compaction:
        ////////////////////////////////////////////////////////////

        // !!! missing !!!
        // implement the prefixSum algorithm
        // 1. Do the reduction step for all scanlines, one scanline per block.
        // 2. Do the reduction step for the last elements of all scanlines, all in one block.
        // 3. Do the spreading step for the last elements of all scanlines, all in one block.
        //    -> The last elements / elements before the scanlines have the right values now.
        // 4. Do the spreading step for all scanlines, one scanline per block.

        // Make sure that gpuFeatureList is filled according to the CPU implementation
        // and that nFeatures has the correct value!
    }

    // now compute the Voronoi Diagram around the detected features.
    voronoiKernel<<<blockGrid, threadBlock, nFeatures * sizeof(int)>>>(gpuVoronoiImg, w, h,
                                                                       nFeatures, gpuFeatureList);

    // download final voronoi image.

    hipMemcpy(cpuImage, gpuVoronoiImg, nPix * 3 * sizeof(float), hipMemcpyDeviceToHost);
    // Write to disk
    writePPM(outName.c_str(), w, h, (float*)cpuImage);

    // Cleanup
    hipDestroyTextureObject(tex);
    hipFreeArray(gpuTex);
    hipFree(gpuFeatureList);
    hipFree(gpuFeatureImg);
    hipFree(gpuPrefixSumShifted);
    hipFree(gpuVoronoiImg);

    delete[] cpuImage;
    delete[] img4;

    checkCUDAError("end of program");

    printf("done\n");
}

void checkCUDAError(const char* msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}
